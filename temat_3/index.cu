#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void myIndex(){
    auto const tidx = threadIdx.x;
    auto const tidy = threadIdx.y;
    auto const bidxx = blockIdx.x;
    auto const bidyy = blockIdx.y;
    auto const idxx = blockIdx.x * blockDim.x + threadIdx.x;
    auto const idxy = blockIdx.y * blockDim.y + threadIdx.y;

    auto const row = blockIdx.y * blockDim.y + threadIdx.y;
    auto const col = blockIdx.x * blockDim.x + threadIdx.x;

    auto const fidx = row * blockDim.x * gridDim.x + col;
    printf("bidxx: %d, bidxy: %d, tidx: %d, tidy: %d, idxx: %d, idxy: %d, row: %d, col: %d, fidx: %d\n",
     bidxx, bidyy, tidx, tidy, idxx, idxy, row, col, fidx);
}

int main(){
    dim3 threads_in_block = {2, 2};
    dim3 blocks = {2, 2};

    myIndex<<<blocks, threads_in_block>>>();
    hipDeviceSynchronize();
}