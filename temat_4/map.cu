#include <hip/hip_runtime.h>
#include <cstdio>
#include <algorithm>
#include <numeric>

// __host__ - dane są dostępne w rapach CPU
// __device__ - dane są dostępne w ramach GPU

template<typename T, typename Func>
__global__ void map(T* const destination, const T* const source, const size_t size,Func func){
    auto const idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size){
        destination[idx] = func(source[idx]);
    }
}

template<typename T>
void showData(const T* const data, const size_t size){
    std::for_each(data, data + size, [](int a){printf("%3d ", a);});
    putchar('\n');
}

int main(){
    using DataType_t = int;
    constexpr static size_t Elements = 128;
    constexpr static size_t MemElements = Elements * sizeof(DataType_t);
    constexpr static size_t ThreadsInBlock = 4;
    constexpr static size_t Block = (Elements + ThreadsInBlock - 1) / ThreadsInBlock;

    DataType_t* hSource = static_cast<DataType_t *>(malloc(MemElements));
    DataType_t* hDestination = static_cast<DataType_t *>(malloc(MemElements));

    DataType_t* dSource = nullptr;
    DataType_t* dDestination = nullptr;

    hipMalloc(&dSource, MemElements);
    hipMalloc(&dDestination, MemElements);

    std::iota(hSource, hSource + Elements, 0);

    showData(hSource, Elements);

    hipMemcpy(dSource, hSource, MemElements, hipMemcpyHostToDevice);

    map<<<Block, ThreadsInBlock>>>(dDestination, dSource, Elements, [] __device__ (auto a) { return 8 * a;});

    hipMemcpy(hDestination, dDestination, MemElements, hipMemcpyDeviceToHost);

    showData(hDestination, Elements);

    hipFree(dDestination);
    hipFree(dSource);

    free(hDestination);
    free(hSource);


    hipDeviceSynchronize();
}